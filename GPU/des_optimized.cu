#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <cmath>
#include <math.h>
#include <math.h>
#include <chrono>
#include <thread>
#include <vector>
#include <fstream>
#include <mutex>
using namespace std;
using namespace std::chrono;

#define sizes 5000// size of side of the game map

#define event_power 5000 // coefficient of destruction
#define x_event 3//x-coordinates of the event
#define y_event 3 //y-coordinates of the event

#define verbose 0 //to print to the console more information

#define print 0//0 -> nothing, 1 -> devastation, 2 -> solidity, 3-> distance 
#define debug 0 // to print debug info
#define print_file 0
#define compact_matrix 1 // this to compact the matrix in order to visualized it better

//usefull for errore handling
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__ __constant__ int gx;
__device__ __constant__ int gy;

struct Block {
	char solidity; // rappresents the capacity of the block to resist [0,10]
	char devastation; // rappresents the effects of the strike [0,10]
	Block() { solidity = 0; devastation = 0; }; //constructor
};

struct Simulation {
	int next_block;
	Block* map;
	mutex mtx;
	int gx; // x-coordinate of ground zero
	int gy; // y-coordinate of ground zero
	Simulation() { gx = 0; gy = 0; map = NULL; next_block = 0; };
};

void print_mat(Simulation* sim){
	if(print_file){
		ofstream f;
		f.open("performance.txt",ios::trunc);
		for (int i = 0; i < sizes * sizes; i++) {
			if ( i % sizes == 0 && i != 0)
				f <<  endl;
			f << sim->map[i].devastation << " ";
		}
		f.close();
	}
	if(compact_matrix){
		ofstream f;
		f.open("matrix_compatted.txt",ios::trunc);
		//for printing the matrix in compact way
		int valore=0;
		int dimensione_sotto_matrice = 100;
		int dimensione_matrice = sizes;

		for(int row =0; row<dimensione_matrice; row+=dimensione_sotto_matrice){
			for(int col=0; col<dimensione_matrice; col += dimensione_sotto_matrice){
				for(int i=row; i<row+dimensione_sotto_matrice;i++){
					for(int j=col; j<col+dimensione_sotto_matrice; j++){
						if((i*sizes+j) <=  (sizes*sizes))
							valore += sim->map[i*sizes+j].devastation;
					}
				}
				float valore_new = valore/(dimensione_sotto_matrice*dimensione_sotto_matrice);
				f << round(valore_new) ;
				f << " ";
				valore = 0;
			}
			f << "\n";
			
		}
		f.close();
	}
	if(!verbose)
		return;
	for ( int i = 0; i < sizes * sizes ; i++) {
		if ( i % sizes == 0 && i != 0)
			cout <<  endl;
		cout << sim->map[i].devastation << " ";
	}
}


__global__ void DES(Block* map, int blocchi, int thread) {

	int id_x = threadIdx.x + blockIdx.x * blockDim.x;
    int num_totale_thread = blocchi*thread;
    int slice = sizes*sizes / num_totale_thread +1 ;

		for(int i = 0 ; i<slice;i++){
            
            int index = id_x * slice +i;
            int row = index / sizes;
	        int col = index - (index/sizes)*sizes;

            if (row < sizes && col < sizes) {
                __syncthreads();
				float xx = __powf(__fsub_rn(row, gx), 2);
        		float y = __fsub_rn(col, gy);
				float sum_xx_yy = __fmaf_rn(y, y , xx);
				int distance_evaluation = floor(__fsqrt_rn(sum_xx_yy));

        		int evaluation = ((event_power - (int)map[row * sizes + col].solidity) / distance_evaluation);
        		char destr_eval_var = 0;
        		if (evaluation > 0 )
        			if(evaluation > 10)
        				destr_eval_var = 9;
        			else
        				destr_eval_var = (char)evaluation;
        		else
        			destr_eval_var = 0;
        		map[row * sizes + col].devastation = destr_eval_var;
        	}
		}

}

void initialization(Simulation* sim) {
	for ( int i = 0; i < sizes*sizes; i++)
		sim->map[i].devastation = 0;
}


int main(int argc, char *argv[]) {

    int num_blocchi = stoi(argv[1]);
    int num_thread = stoi(argv[2]);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEvent_t start_simulation, stop_simulation;
	hipEventCreate(&start_simulation);
	hipEventCreate(&stop_simulation);

	ofstream f;
	f.open("performance.txt",ios::app);

	srand(2);
	Simulation sim;
	sim.map = new Block[sizes*sizes];

	for (int i = 0; i < sizes; i++)
		for (int j = 0; j < sizes; j++)
			sim.map[i*sizes+j].solidity = rand() % 10;

	sim.next_block = 0;

	int* gx_host = (int*)malloc(sizeof(int));
	*gx_host = x_event;
	int* gy_host = (int*)malloc(sizeof(int));
	*gy_host = y_event;

	
	long size_map = sizeof(Block) * sizes * sizes;
	Block* copy_map;

	hipEventRecord(start);

	hipMemcpyToSymbol(HIP_SYMBOL(gx),gx_host,sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(gy),gy_host,sizeof(int));
	
	hipMalloc((void**)&copy_map, size_map);
	steady_clock::time_point begin = steady_clock::now();
	hipMemcpy(copy_map, sim.map, size_map, hipMemcpyHostToDevice);
	steady_clock::time_point end = steady_clock::now();

    hipEventRecord(start_simulation);
	DES <<<  num_blocchi, num_thread >>> (copy_map,num_blocchi,num_thread);
	hipEventRecord(stop_simulation);

	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
	
	hipMemcpy(sim.map, copy_map, size_map, hipMemcpyDeviceToHost);

	hipEventRecord(stop);
	hipEventSynchronize(stop);

	print_mat(&sim);
    cout<<endl;

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	cout <<  setprecision(7) << "Total simulation time " << milliseconds/pow(10,3);

	hipEventElapsedTime(&milliseconds, start_simulation, stop_simulation);
	cout << endl << setprecision(7) << "Simulation time " << milliseconds/pow(10,3);

	cout << endl << "Copy device to host time: " << duration_cast<microseconds>(end - begin).count() / pow(10, 6)<<endl;
		
	f<<	milliseconds/pow(10,3)<< " ("<<num_blocchi<<","<<num_thread<<")"<<endl;
    f.close();
	
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipEventDestroy(start_simulation);
	hipEventDestroy(stop_simulation);

	return 0;
}